#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>

#define SUBMATRIX_SIZE 10000
#define BLOCK_SIZE 16

float getnum() {
  return ((float) rand())/((float) RAND_MAX);
}


__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
  __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * BLOCK_SIZE + ty;
  int Col = bx * BLOCK_SIZE + tx;
  float Pvalue = 0;
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int m = 0; m < Width/BLOCK_SIZE; ++m) {
    // Collaborative loading of Md and Nd tiles into shared memory
    Mds[ty][tx] = Md[Row*Width + (m*BLOCK_SIZE + tx)];
    Nds[ty][tx] = Nd[Col + (m*BLOCK_SIZE + ty)*Width];
    __syncthreads();
    for (int k = 0; k < BLOCK_SIZE; ++k)
      Pvalue += Mds[ty][k] * Nds[k][tx];
    __syncthreads();
  }
  Pd[Row*Width+Col] = Pvalue;
}


void make_identity(float* a, int startx, int starty, int length, int n) {
  // fill a with the identity from start to end
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = (i-startx==j-starty) ? 1.0f : 0.0f;
    }
  }
}

void make_negidentity(float* a, int startx, int starty, int length, int n) {
  // fill a with the identity from start to end
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = (i-startx==j-starty) ? -1.0f : 0.0f;
    }
  }
}

void make_x(float* x, int length) {
  int i, j;
  for (i=0; i<length; i++) {
    for (j=0; j<length; j++) {
      x[i*length+j] = getnum();
      //x[i*length+j] = 0.3;
    }
  }
}

void make_zero(float* a, int startx, int starty, int length, int n) {
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = 0.0f;
    }
  }
}

void copy_x(float* a, float* x, int startx, int starty, int length, int n) {
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = x[(i-startx)*length+(j-starty)];
    }
  }
}

void copy_2x(float* a, float* x, int startx, int starty, int length, int n) {
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = 2*x[(i-startx)*length+(j-starty)];
    }
  }
}

void copy_negx(float* a, float* x, int startx, int starty, int length, int n) {
  int i, j;
  for (i=startx; i<length+startx; i++) {
    for (j=starty; j<length+starty; j++) {
      a[i*n+j] = (-1)*x[(i-startx)*length+(j-starty)];
    }
  }
}

void make_result(float* a, int length) {
  int i, j;
  int half = length>>1;
  for (i=0; i<length; i++) {
    for (j=0; j<length; j++) {
      if (i == j) {
	if (i>=half)
	  a[i*length+j] = -(1.0f);
	else
	  a[i*length+j] = 1.0f;
      }
      else
	a[i*length+j] = 0.0f;
    }
  }
}

#define ffabs(val) (val) < 0.0f ? (-(val)) : (val)

float rothVerf(float* a, float* b, int n) {
  float sum = 0;
  int i, j;
  for (i=0; i<n; i++) {
    for (j=0; j<n; j++) {
      sum += ffabs(a[i*n+j] - b[i*n+j]);
    }
  }
  return sum;
}

void print_mat(float* a, int n) {
  int i, j;
  if (n<64) {
    for (i=0; i<n; i++) {
      for (j=0; j<n; j++) {
	printf("%.3f\t", a[i*n+j]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

float trace(float* a, int n) {
  int i;
  float total = 1.0f;
  for (i=0; i<n; i++) {
    total *= a[i*n+i];
  }
  return total;
}


int main() {
  srand(100);
  int n = 2*SUBMATRIX_SIZE;
  int half = SUBMATRIX_SIZE;
  size_t totalsize = sizeof(float)*n*n;
  size_t halfsize = sizeof(float)*half*half;
  float *x, *a, *b, *c, *d;

  hipHostMalloc((void**) &a, totalsize);
  hipHostMalloc((void**) &b, totalsize);
  hipHostMalloc((void**) &c, totalsize);
  hipHostMalloc((void**) &d, totalsize);
  hipHostMalloc((void**) &x, halfsize);
  
  if ((x==NULL) || (a==NULL) || (b==NULL) || (c==NULL) ||
      (d==NULL)) {
    printf("Matrix allocation error on host\n");
    exit(1);
  }

  make_x(x, half);
  print_mat(x, half);

  // construct first matrix
  make_identity(a, 0, 0, half, n);
  copy_x(a, x, 0, half, half, n);
  make_zero(a, half, 0, half, n);
  make_identity(a, half, half, half, n);
  printf("Trace of a: %f\n", trace(a, n));

  // second matrix
  make_identity(b, 0, 0, half, n);
  copy_2x(b, x, 0, half, half, n);
  make_zero(b, half, 0, half, n);
  make_negidentity(b, half, half, half, n);

  // third
  make_identity(c, 0, 0, half, n);
  copy_negx(c, x, 0, half, half, n);
  make_zero(c, half, 0, half, n);
  make_identity(c, half, half, half, n);

  // result
  make_result(d, n);
  print_mat(a, n);
  print_mat(b, n);
  print_mat(c, n);
  print_mat(d, n);

  // allocate on device
  float *dev_a, *dev_b, *dev_c, *dev_inter;
  hipMalloc((void**) &dev_a, totalsize);
  hipMalloc((void**) &dev_b, totalsize);
  hipMalloc((void**) &dev_c, totalsize);
  hipMalloc((void**) &dev_inter, totalsize);
  
  // copy to device
  hipMemcpy(dev_a, a, totalsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, totalsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, totalsize, hipMemcpyHostToDevice);

  unsigned int grid_rows = n / BLOCK_SIZE;
  unsigned int grid_cols = n / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // intermediate matrix product
  //MatrixMulKernel<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_inter, n);
  hipblasHandle_t handle;
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, dev_a, n, dev_b, n, &beta, dev_inter, n);
  hipDeviceSynchronize();

  // reuse old matrix
  //MatrixMulKernel<<<dimGrid, dimBlock>>>(dev_inter, dev_c, dev_a, n);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &alpha, dev_inter, n, dev_c, n, &beta, dev_a, n);

  // bring product back to cpu
  hipMemcpy(a, dev_a, totalsize, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // check a against the result d
  float sum = rothVerf(a, d, n);
  printf("Total Error: %f\n", sum);
  print_mat(a, n);

  // cleanup and exit
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipFree(dev_inter);
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  hipHostFree(d);
  hipHostFree(x);
  return 0;
}
